#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "utils/utils.h"

// ============================================================================

__global__ void add_arrays(const float* a, const float* b, float* c, int N)
{
	unsigned i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < N) c[i] = a[i] + b[i];
}

// ============================================================================

int main(void)
{
	hipSetDevice(0);

	int N = 1 << 20;
	const size_t block_size = 256;
	size_t num_blocks = N / block_size;
    if (N % block_size)
		++num_blocks;

	float* host_array_a = 0;
	float* host_array_b = 0;
	float* host_array_c = 0;

	float* device_array_a = 0;
	float* device_array_b = 0;
	float* device_array_c = 0;

	// ------------------------------------------------------------------------

	host_array_a = (float*)malloc(N * sizeof(int));
	host_array_b = (float*)malloc(N * sizeof(int));
	host_array_c = (float*)malloc(N * sizeof(int));

	// ------------------------------------------------------------------------

	hipMalloc(&device_array_a, N * sizeof(int));
    hipMalloc(&device_array_b, N * sizeof(int));
	hipMalloc(&device_array_c, N * sizeof(int));

	// ------------------------------------------------------------------------

	fill_array_float(host_array_a, N);
	fill_array_random(host_array_b, N);

	// ------------------------------------------------------------------------

	hipMemcpy(device_array_a, host_array_a, N * sizeof(int),
                hipMemcpyHostToDevice);

	hipMemcpy(device_array_b, host_array_b, N * sizeof(int),
                hipMemcpyHostToDevice);

	// ------------------------------------------------------------------------

	add_arrays<<<num_blocks, block_size>>>(device_array_a, device_array_b,
		                                    device_array_c, N);
	hipDeviceSynchronize();

	// ------------------------------------------------------------------------

	hipMemcpy(host_array_c, device_array_c, N * sizeof(int),
                hipMemcpyDeviceToHost);

	check_task_2(host_array_a, host_array_b, host_array_c, N);

	// ------------------------------------------------------------------------

	free(host_array_a);
	free(host_array_b);
	free(host_array_c);

	hipFree(device_array_a);
    hipFree(device_array_b);
	hipFree(device_array_c);

	return 0;
}

// ============================================================================