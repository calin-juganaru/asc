#include <stdio.h>
#include <math.h>
#include "utils/utils.h"

#define N 1000000
#define BUF_2M	( 2 * 1024 * 1024)
#define BUF_32M	(32 * 1024 * 1024)

// ============================================================================

int main(void)
{
	hipSetDevice(0);

	int* host_array_a = 0;
	int* host_array_b = 0;

	int* device_array_a = 0;
	int* device_array_b = 0;
	int* device_array_c = 0;


	// ------------------------------------------------------------------------

	host_array_a = (int*)malloc(BUF_32M * sizeof(int));
	host_array_b = (int*)malloc(BUF_32M * sizeof(int));

	// ------------------------------------------------------------------------

	hipMalloc(&device_array_a, BUF_32M * sizeof(int));
	hipMalloc(&device_array_b, BUF_32M * sizeof(int));
	hipMalloc(&device_array_c, BUF_2M  * sizeof(int));

	// ------------------------------------------------------------------------

	if (host_array_a == 0 || host_array_b == 0 ||
		device_array_a == 0 || device_array_b == 0 ||
		device_array_c == 0)
	{
		printf("[*] Error!\n");
		return 1;
	}

	for (int i = 0; i < BUF_32M; ++i)
	{
		host_array_a[i] = i % 32;
		host_array_b[i] = i % 2;
	}

	printf("Before the swap:\n a[i]    b[i]\n");
	for (int i = 0; i < N; ++i)
		printf("  %d       %d\n", host_array_a[i], host_array_b[i]);

	// ------------------------------------------------------------------------

	hipMemcpy(device_array_a, host_array_a,
                BUF_32M * sizeof(int),
                hipMemcpyHostToDevice);

	hipMemcpy(device_array_b, host_array_b,
		        BUF_32M * sizeof(int),
                hipMemcpyHostToDevice);

	// ------------------------------------------------------------------------

	for (int i = 0; i != BUF_32M; i += BUF_2M)
	{
		hipMemcpy(device_array_c, device_array_b + i,
			        BUF_2M * sizeof(int),
			        hipMemcpyDeviceToDevice);

		hipMemcpy(device_array_b + i, device_array_a + i,
			        BUF_2M * sizeof(int),
			        hipMemcpyDeviceToDevice);

		hipMemcpy(device_array_a + i, device_array_c,
			        BUF_2M * sizeof(int),
			        hipMemcpyDeviceToDevice);
	}

	// ------------------------------------------------------------------------

	hipMemcpy(host_array_a, device_array_a,
		        BUF_32M * sizeof(int),
                hipMemcpyDeviceToHost);

	hipMemcpy(host_array_b, device_array_b,
		        BUF_32M * sizeof(int),
                hipMemcpyDeviceToHost);

	printf("\nAfter the swap:\n a[i]    b[i]\n");
	for (int i = 0; i < N; ++i)
		printf("  %d       %d\n", host_array_a[i], host_array_b[i]);

	// ------------------------------------------------------------------------

	free(host_array_a);
	free(host_array_b);

	hipFree(device_array_a);
	hipFree(device_array_b);
	hipFree(device_array_c);

	return 0;
}

// ============================================================================